#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
// --------------------------------------------------------------------
#define CHECK_CUDA(call)                                            \
  {                                                                 \
    hipError_t err = call;                                         \
    if (err != hipSuccess) {                                       \
      std::cerr << "CUDA error in " << __FILE__ << " at line "      \
            << __LINE__ << ": " << hipGetErrorString(err) << "\n"; \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  }

#define CHECK_CUBLAS(call)                                          \
  {                                                                 \
    hipblasStatus_t err = call;                                      \
    if (err != HIPBLAS_STATUS_SUCCESS) {                             \
      std::cerr << "cuBLAS error in " << __FILE__ << " at line "    \
            << __LINE__ << ": " << err << "\n";                     \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  }
// --------------------------------------------------------------------
void solveBatchedLinearSystems(int32_t n, int32_t m, double* As, double* bs, double* xs) {
  double *As_, *bs_;
  double **As_array_, **bs_array_;
  int32_t *pivots_, *info_;

  hipblasHandle_t handle_;
  CHECK_CUBLAS(hipblasCreate(&handle_));

  // Allocate GPU memory
  CHECK_CUDA(hipMalloc((void**)&As_, m * n * n * sizeof(double)));
  CHECK_CUDA(hipMalloc((void**)&bs_, m * n * sizeof(double)));
  CHECK_CUDA(hipMalloc((void**)&pivots_, m * n * sizeof(int32_t)));
  CHECK_CUDA(hipMalloc((void**)&info_, m * sizeof(int32_t)));

  // Copy data from pinned host memory to device memory
  CHECK_CUDA(hipMemcpy(As_, As, m * n * n * sizeof(double), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(bs_, bs, m * n * sizeof(double), hipMemcpyHostToDevice));

  // Allocate and set up batched matrix pointers
  CHECK_CUDA(hipMalloc((void**)&As_array_, m * sizeof(double*)));
  CHECK_CUDA(hipMalloc((void**)&bs_array_, m * sizeof(double*)));

  double **As_array, **bs_array;
  CHECK_CUDA(hipHostMalloc((void**)&As_array, m * sizeof(double*), hipHostMallocDefault)); // Pinned memory for faster transfer
  CHECK_CUDA(hipHostMalloc((void**)&bs_array, m * sizeof(double*), hipHostMallocDefault)); // Pinned memory for faster transfer

  for (int32_t i = 0; i < m; i++) {
    As_array[i] = As_ + i * n * n;
    bs_array[i] = bs_ + i * n;
  }

  CHECK_CUDA(hipMemcpy(As_array_, As_array, m * sizeof(double*), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(bs_array_, bs_array, m * sizeof(double*), hipMemcpyHostToDevice));

  // Perform LU factorization
  CHECK_CUBLAS(hipblasDgetrfBatched(handle_, n, As_array_, n, pivots_, info_, m));

  // Solve Ax = b using LU factors
  CHECK_CUBLAS(hipblasDgetrsBatched(handle_, HIPBLAS_OP_N, n, 1, As_array_, n, pivots_, bs_array_, n, info_, m));

  // Copy solution from device to pinned host memory
  CHECK_CUDA(hipMemcpy(xs, bs_, m * n * sizeof(double), hipMemcpyDeviceToHost));

  // Free GPU memory
  hipFree(As_);
  hipFree(bs_);
  hipFree(pivots_);
  hipFree(info_);
  hipFree(As_array_);
  hipFree(bs_array_);

  // Free pinned host memory
  hipHostFree(As_array);
  hipHostFree(bs_array);

  hipblasDestroy(handle_);
}
// --------------------------------------------------------------------
int32_t main() {
  int32_t n = 3; // Matrix size
  int32_t m = 2; // Number of systems

  double *As, *bs, *xs;

  // Allocate pinned host memory
  CHECK_CUDA(hipHostMalloc((void**)&As, m * n * n * sizeof(double), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc((void**)&bs, m * n * sizeof(double), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc((void**)&xs, m * n * sizeof(double), hipHostMallocDefault));

  // Initialize A and B
  double A_data[] = {
    4, 1, 2,   // System 1: A
    1, 3, 2,
    2, 2, 3,

    5, 2, 1,   // System 2: A
    2, 6, 3,
    1, 3, 7
  };

  double B_data[] = {
    1, 2, 3,  // System 1: b
    4, 5, 6   // System 2: b
  };

  memcpy(As, A_data, m * n * n * sizeof(double));
  memcpy(bs, B_data, m * n * sizeof(double));

  // Solve the linear systems
  solveBatchedLinearSystems(n, m, As, bs, xs);

  // Print solutions
  std::cout << "Solutions (X):\n";
  for (int32_t i = 0; i < m; i++) {
    std::cout << "System " << i + 1 << ": ";
    for (int32_t j = 0; j < n; j++) {
      std::cout << xs[i * n + j] << " ";
    }
    std::cout << "\n";
  }

  // Free pinned host memory
  hipHostFree(As);
  hipHostFree(bs);
  hipHostFree(xs);

  return 0;
}
// --------------------------------------------------------------------