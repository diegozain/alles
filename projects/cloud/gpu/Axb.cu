#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
// --------------------------------------------------------------------
// 
// 🦠💻🧠
// ** pinned memory **
// double *As;
// cudaMallocHost(&As, m * n * n * sizeof(double));
// cudaFreeHost(As);
// 
// 💻🧠
// ** heap memory **
// double* As = (double*)malloc(m * n * n *  sizeof(double));
// free(As);
// 
// --------------------------------------------------------------------
// 🔵🔵 windows 🔵🔵
// nvcc -o Axb.exe Axb.cu -I. -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.6\include" -L"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.6\lib\x64" -lcudart -lcublas --gpu-architecture=sm_75 --expt-relaxed-constexpr -O3 -dlto -use_fast_math
// --------------------------------------------------------------------
void Axbsolver(int32_t n, int32_t m, double* As, double* bs, double* xs) {
  // ⌚
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  // --------------------------------------------------------------------
  // 🐦‍⬛🧠
  hipblasHandle_t handle_;
  hipblasCreate(&handle_);

  double *As_, *bs_;
  double **As_array_, **bs_array_;
  int32_t *info_; // for cublasDgetrfBatched
  int32_t *pivots_;

  hipMalloc(&As_, m * n * n * sizeof(double));
  hipMalloc(&bs_, m * n * sizeof(double));
  hipMalloc(&As_array_, m * sizeof(double*));
  hipMalloc(&bs_array_, m * sizeof(double*));
  hipMalloc(&info_, m * sizeof(int32_t));
  hipMalloc(&pivots_, m * n * sizeof(int32_t));
  // --------------------------------------------------------------------
  // 🦠💻🧠 pinned memory
  double **As_array, **bs_array;
  hipHostMalloc(&As_array, m * sizeof(double*), hipHostMallocDefault);
  hipHostMalloc(&bs_array, m * sizeof(double*), hipHostMallocDefault);
  int32_t *info; // for cublasDgetrsBatched
  hipHostMalloc(&info, m * sizeof(int32_t*), hipHostMallocDefault);
  // --------------------------------------------------------------------
  // 🦠 ⟵ 💻
  hipMemcpy(As_, As, m * n * n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(bs_, bs, m * n * sizeof(double), hipMemcpyHostToDevice);

  // 🐦‍⬛ ⟵ 💻
  for (int32_t i = 0; i < m; i++) {
    As_array[i] = As_ + i * n * n;
    bs_array[i] = bs_ + i * n;
  }
  hipMemcpy(As_array_, As_array, m * sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(bs_array_, bs_array, m * sizeof(double*), hipMemcpyHostToDevice);
  // --------------------------------------------------------------------
  // 🚀🐦‍⬛
  
  // ∘∘ LU ∘∘
  hipblasDgetrfBatched(handle_, n, As_array_, n, pivots_, info_, m);
  hipblasDgetrsBatched(handle_, HIPBLAS_OP_N, n, 1, As_array_, n, pivots_, bs_array_, n, info, m);
  // --------------------------------------------------------------------
  // 💻 ⟵ 🐦‍⬛
  hipMemcpy(xs, bs_, m * n * sizeof(double), hipMemcpyDeviceToHost);
  // --------------------------------------------------------------------
  // 🚿🦠🐦‍⬛
  hipblasDestroy(handle_);
  
  hipFree(As_);
  As_ = nullptr;
  hipFree(bs_);
  bs_ = nullptr;
  hipFree(info_);
  info_ = nullptr;
  hipFree(As_array_);
  As_array_ = nullptr;
  hipFree(bs_array_);
  bs_array_ = nullptr;
  hipFree(pivots_);
  pivots_ = nullptr;

  // 🚿🦠🐦‍⬛💻
  hipHostFree(As_array);
  hipHostFree(bs_array);
  hipHostFree(info);
  // --------------------------------------------------------------------
  // ⌚
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  // ⌚🚿
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "  Axbsolver elapsed time: " << elapsedTime/1e3 << " seconds" << "\n";
}
// --------------------------------------------------------------------
int32_t main() {
  // 🧠
  int32_t n = 5;
  int32_t m = 3;

  // 🦠💻🧠 pinned memory
  double *As, *bs, *xs;
  hipHostMalloc(&As, m * n * n * sizeof(double), hipHostMallocDefault);
  hipHostMalloc(&bs, m * n * sizeof(double), hipHostMallocDefault);
  hipHostMalloc(&xs, m * n * sizeof(double), hipHostMallocDefault);

  // 🖊️
  double* As_data = (double*)malloc(m * n * n *  sizeof(double));
  double* bs_data = (double*)malloc(m * n *  sizeof(double));
  for (int32_t i = 0; i < m * n * n; i++) {
    As_data[i] = static_cast<double>(rand() % 10);
  }
  for (int32_t i = 0; i < m * n; i++) {
    bs_data[i] = static_cast<double>(rand() % 10);
  }

  memcpy(As, As_data, m * n * n * sizeof(double));
  memcpy(bs, bs_data, m * n * sizeof(double));

  // 🖨️
  std::cout << "\n\n ::: solving now :::" << "\n\n";
  for (int32_t j = 0; j < m; j++) {
    std::cout << "A" << j + 1 << ": \n";
    for (int32_t ii = 0; ii < n; ii++) {
      for (int32_t i = 0; i < n; i++) {
        std::cout << As[j*n*n + ii * n + i] << " ";
      }
      std::cout << "\n";
    }
    std::cout << "\n";
  }
  for (int32_t i = 0; i < m; i++) {
    std::cout << "bs" << i + 1 << ": ";
    for (int32_t j = 0; j < n; j++) {
      std::cout << bs[i * n + j] << " ";
    }
    std::cout << "\n";
  }
  std::cout << "\n\n";

  // 🚀🐦‍⬛
  Axbsolver(n, m, As, bs, xs);

  // 🖨️
  std::cout << "\n\n ::: result :::" << "\n\n";
  for (int32_t i = 0; i < m; i++) {
    std::cout << "xs" << i + 1 << ": ";
    for (int32_t j = 0; j < n; j++) {
      std::cout << xs[i * n + j] << " ";
    }
    std::cout << "\n";
  }

  // 🚿🦠💻
  hipHostFree(As);
  hipHostFree(bs);
  hipHostFree(xs);

  // 🚿
  free(As_data);
  As_data = nullptr;
  free(bs_data);
  bs_data = nullptr;

  return 0;
}
// --------------------------------------------------------------------