// --------------------------------------------------------------------
// m systems of equations of the form:
//      Ax=b, A of size n×n
// 
// * store in memory
//   all As and bs in continuous arrays
// 
// * factorize
//   cholesky (positive definite?)
//     cublasDpotrfBatched()
//   LU
//     hipblasDgetrfBatched()
// 
// * solve 
//   cholesky
//     cublasDpotrsBatched()
//   LU
//     hipblasDgetrsBatched()
// 
// * retreive solution
// 
// * clean gpu
// --------------------------------------------------------------------
// 🐦‍⬛🧠
double* As_; // Batched matrix A
double* bs_; // Batched RHS vector B
int32_t* info_; // Status info for factorization
double** As_array_;
double** bs_array_;

hipMalloc((void**)&As_, m * n * n * sizeof(double));
hipMalloc((void**)&bs_, m * n * sizeof(double));
hipMalloc((void**)&info_, m * sizeof(int32_t));
hipMalloc((void**)&As_array_, m * sizeof(double*));
hipMalloc((void**)&bs_array_, m * sizeof(double*));

double **As_array, **bs_array;
hipHostMalloc((void**)&As_array, m * sizeof(double*));
hipHostMalloc((void**)&bs_array, m * sizeof(double*)); 

hipblasHandle_t handle_;
hipblasCreate(&handle_);

// 🐦‍⬛ ⟵ 💻
hipMemcpy(As_, As, m * n * n * sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(bs_, bs, m * n * sizeof(double), hipMemcpyHostToDevice);

for (int32_t i = 0; i < m; i++) {
  As_array[i] = As_ + i * n * n;
  bs_array[i] = bs_ + i * n;
}

hipMemcpy(As_array_, As_array, m * sizeof(double*), hipMemcpyHostToDevice);
hipMemcpy(bs_array_, bs_array, m * sizeof(double*), hipMemcpyHostToDevice);

// 🚀🐦‍⬛

// cholesky factorization in batch
cublasDpotrfBatched(handle_, HIPBLAS_FILL_MODE_LOWER, n, As_array_, n, info_, m);
// solve Ax = b using Cholesky factors
cublasDpotrsBatched(handle_, HIPBLAS_FILL_MODE_LOWER, n, 1, As_array_, n, bs_array, n, info_, m);

// LU
int32_t* d_pivots; // Pivot array for LU
hipMalloc((void**)&d_pivots, m * n * sizeof(int32_t));
hipblasDgetrfBatched(handle_, n, As_array_, n, d_pivots, info_, m);
hipblasDgetrsBatched(handle_, HIPBLAS_OP_N, n, 1, As_array_, n, d_pivots, bs_array, n, info_, m);

// 💻 ⟵ 🐦‍⬛
hipMemcpy(xs, bs_, m * n * sizeof(double), hipMemcpyDeviceToHost);

// 🚿🐦‍⬛
hipFree(As_);
hipFree(bs_);
hipFree(info_);
hipFree(As_array_);
hipFree(bs_array);
hipFree(d_pivots);
hipblasDestroy(handle_);




